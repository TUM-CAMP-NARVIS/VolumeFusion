#include <af/LinearSolver.cuh>

namespace af {

hipsolverHandle_t createHandleDn() {
    hipsolverHandle_t handleDn = NULL;
    hipsolverDnCreate(&handleDn);
    CUDA_CHECK;
    return handleDn;
}

bool linSolvCholDn(hipsolverHandle_t handleDn, float* ADense, float* bDense, const int dim) {
    if (ADense == NULL || bDense == NULL)
        return false;
    // throw std::runtime_error("af::linSolvCholDn(): input arrays cannot be empty.");

    Timer timer;

    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
    const int nrhs              = 1;
    const int lda               = dim;
    const int ldb               = dim;

    int workSize = 0;
    hipsolverDnSpotrf_bufferSize(handleDn, uplo, dim, ADense, lda, &workSize);
    CUDA_CHECK;

    thrust::device_vector<float> workspace_d(workSize);
    thrust::device_vector<int> devInfo_d(1, 0);

    // Cholesky Factorization
    hipsolverDnSpotrf(handleDn, uplo, dim, ADense, lda, workspace_d.data().get(), workSize, devInfo_d.data().get());
    if (devInfo_d[0] != 0) {
        std::cout << "af::linSolvCholDn(): hipsolverDnSpotrf failed, devinfo : " + std::to_string(devInfo_d[0]) << "\n";
        return false;
        // throw std::runtime_error("af::linSolvCholDn(): hipsolverDnSpotrf failed, devinfo : " + std::to_string(devInfo_d[0]));
    }
    CUDA_CHECK;

    // Solve Ax = b
    hipsolverDnSpotrs(handleDn, uplo, dim, nrhs, ADense, lda, bDense, ldb, devInfo_d.data().get());
    if (devInfo_d[0] != 0) {
        std::cout << "af::hipsolverDnSpotrs(): hipsolverDnSpotrf failed, devinfo : " + std::to_string(devInfo_d[0]) << "\n";
        return false;
        // throw std::runtime_error("af::linSolvCholDn(): hipsolverDnSpotrs failed, devinfo : " + std::to_string(devInfo_d[0]));
    }
    CUDA_CHECK;

    return true;
}

bool linSolvLUDn(hipsolverHandle_t handleDn, float* ADense, float* bDense, const int dim) {
    if (ADense == NULL || bDense == NULL)
        return false;
    // throw std::runtime_error("af::linSolvLUDn(): input arrays cannot be empty.");
    hipStream_t stream = NULL;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSetStream(handleDn, stream);

    const int nrhs = 1;
    const int lda  = dim;
    const int ldb  = dim;

    int workSize = 0;
    hipsolverDnSgetrf_bufferSize(handleDn, dim, dim, ADense, lda, &workSize);
    hipDeviceSynchronize();
    CUDA_CHECK;

    thrust::device_vector<float> workspace_d(workSize);
    thrust::device_vector<int> devInfo_d(1, 0);

    // LU Factorization
    hipsolverDnSgetrf(handleDn, dim, dim, ADense, lda, workspace_d.data().get(), NULL, devInfo_d.data().get());
    hipDeviceSynchronize();
    if (devInfo_d[0] != 0)
        return false;
    // throw std::runtime_error("af::linSolvLUDn(): hipsolverDnSgetrf failed, devinfo : " + std::to_string(devInfo_d[0]));
    CUDA_CHECK;

    // Solve Ax = b
    hipsolverDnSgetrs(handleDn, HIPBLAS_OP_N, dim, nrhs, ADense, lda, NULL, bDense, ldb, devInfo_d.data().get());
    hipDeviceSynchronize();
    if (devInfo_d[0] != 0)
        throw std::runtime_error("af::linSolvLUDn(): hipsolverDnSgetrs failed, devinfo : " + std::to_string(devInfo_d[0]));
    CUDA_CHECK;

    if (stream)
        hipStreamDestroy(stream);

    return true;
}

bool linSolvQRDn(hipsolverHandle_t handleDn, hipblasHandle_t handleCublas, float* ADense, float* bDense, const int dim) {
    if (ADense == NULL || bDense == NULL)
        return false;

    const int m    = dim;
    const int lda  = m;
    const int ldb  = m;
    const int nrhs = 1;

    thrust::device_vector<float> tau_d(m);

    int workSize = 0;
    hipsolverDnSgeqrf_bufferSize(handleDn, dim, dim, ADense, lda, &workSize);
    hipDeviceSynchronize();
    CUDA_CHECK;

    thrust::device_vector<float> workspace_d(workSize);
    thrust::device_vector<int> devInfo_d(1, 0);

    hipsolverDnSgeqrf(handleDn, dim, dim, ADense, lda, tau_d.data().get(), workspace_d.data().get(), workSize,
                     devInfo_d.data().get());
    hipDeviceSynchronize();
    if (devInfo_d[0] != 0) {
        std::cout << "af::linSolvQRDn(): hipsolverDnSgeqrf failed, devinfo : " + std::to_string(devInfo_d[0]) << "\n";
        return false;
        // throw std::runtime_error("af::linSolvCholDn(): hipsolverDnSpotrf failed, devinfo : " + std::to_string(devInfo_d[0]));
    }
    CUDA_CHECK;

    hipsolverDnSormqr(handleDn, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, m, nrhs, m, ADense, lda, tau_d.data().get(), bDense, ldb,
                     workspace_d.data().get(), workSize, devInfo_d.data().get());
    hipDeviceSynchronize();
    if (devInfo_d[0] != 0) {
        std::cout << "af::linSolvQRDn(): hipsolverDnSormqr failed, devinfo : " + std::to_string(devInfo_d[0]) << "\n";
        return false;
        // throw std::runtime_error("af::linSolvCholDn(): hipsolverDnSpotrf failed, devinfo : " + std::to_string(devInfo_d[0]));
    }
    CUDA_CHECK;

    const float one = 1.f;
    hipblasStrsm(handleCublas, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, nrhs, &one, ADense,
                lda, bDense, ldb);
    hipDeviceSynchronize();

    return true;
}

/*
 *  solve A*x = b by LU with partial pivoting
 *
 */
bool linearSolverLU(hipsolverHandle_t handle, int n, float* A, int lda, float* b, int* ipiv) {
    int bufferSize = 0;
    int* info      = NULL;
    float* buffer  = NULL;
    int h_info     = 0;

    hipsolverDnSgetrf_bufferSize(handle, n, n, (float*)A, lda, &bufferSize);

    hipMalloc(&info, sizeof(int));
    hipMalloc(&buffer, sizeof(float) * bufferSize);

    hipMemset(info, 0, sizeof(int));

    // getrf will overwrite A with L
    hipsolverDnSgetrf(handle, n, n, A, lda, buffer, ipiv, info);
    hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost);

    if (0 != h_info) {
        fprintf(stderr, "Error: LU factorization failed\n");
        return false;
    }

    hipsolverDnSgetrs(handle, HIPBLAS_OP_N, n, 1, A, lda, ipiv, b, n, info);
    hipDeviceSynchronize();

    if (info) {
        hipFree(info);
    }
    if (buffer) {
        hipFree(buffer);
    }

    return true;
}

bool linearSolverLUStable(hipsolverHandle_t handle, int n, const float* Acopy, int lda, const float* b, int* ipiv, float* x) {
    int bufferSize = 0;
    int* info      = NULL;
    float* buffer  = NULL;
    float* A       = NULL;
    // int* ipiv      = NULL;  // pivoting sequence
    int h_info     = 0;

    hipsolverDnSgetrf_bufferSize(handle, n, n, (float*)Acopy, lda, &bufferSize);

    hipMalloc(&info, sizeof(int));
    hipMalloc(&buffer, sizeof(float) * bufferSize);
    hipMalloc(&A, sizeof(float) * lda * n);
    hipMalloc(&ipiv, sizeof(int) * n);

    // prepare a copy of A because getrf will overwrite A with L
    hipMemcpy(A, Acopy, sizeof(float) * lda * n, hipMemcpyDeviceToDevice);
    hipMemset(info, 0, sizeof(int));

    hipsolverDnSgetrf(handle, n, n, A, lda, buffer, ipiv, info);
    hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost);

    if (0 != h_info) {
        fprintf(stderr, "Error: LU factorization failed\n");
        return false;
    }

    hipMemcpy(x, b, sizeof(float) * n, hipMemcpyDeviceToDevice);
    hipsolverDnSgetrs(handle, HIPBLAS_OP_N, n, 1, A, lda, ipiv, x, n, info);
    hipDeviceSynchronize();

    if (info) {
        hipFree(info);
    }
    if (buffer) {
        hipFree(buffer);
    }
    if (A) {
        hipFree(A);
    }
    // if (ipiv) {
    //     hipFree(ipiv);
    // }

    return true;
}

}  // namespace af